#define CUSP_USE_TEXTURE_MEMORY

#include <cusp/csr_matrix.h>
#include <cusp/io/matrix_market.h>
#include <cusp/gallery/poisson.h>

#include <iostream>
#include <string>
#include <map>
#include <cmath>
#include <limits>

#include <cusp/multiply.h>
#include <cusp/detail/device/spmv/csr_scalar.h>

#include "bytes_per_spmv.h"
#include "utility.h"
#include "benchmark.h"

#include "nitro_config.h"
#include <nitro.h>

#include "variants.h"
#include "features.h"
#include "constraints.h"

typedef std::map<std::string, std::string> ArgumentMap;
ArgumentMap args;

std::string process_args(int argc, char ** argv)
{
    std::string filename;

    for(int i = 1; i < argc; i++)
    {
        std::string arg(argv[i]);

        if (arg.substr(0,2) == "--")
        {   
            std::string::size_type n = arg.find('=',2);

            if (n == std::string::npos)
                args[arg.substr(2)] = std::string();              // (key)
            else
                args[arg.substr(2, n - 2)] = arg.substr(n + 1);   // (key,value)
        }
        else
        {
            filename = arg;
        }
    }

    return filename;
}

void usage(int argc, char** argv)
{
    std::cout << "Usage:\n";
    std::cout << "\t" << argv[0] << "\n";
    std::cout << "\t" << argv[0] << " my_matrix.mtx\n";
    std::cout << "\t" << argv[0] << " my_matrix.mtx --device=1\n";
    std::cout << "\t" << argv[0] << " my_matrix.mtx --value_type=double\n\n";
    std::cout << "Note: my_matrix.mtx must be real-valued sparse matrix in the MatrixMarket file format.\n"; 
    std::cout << "      If no matrix file is provided then a simple example is created.\n";
}

template <typename IndexType, typename ValueType>
void test_all_formats(std::string& filename)
{
    int device_id  = args.count("device") ? atoi(args["device"].c_str()) :  0;
    set_device(device_id);
    list_devices();

    std::cout << "Running on Device " << device_id << "\n\n";
    
    // load a matrix stored in MatrixMarket format
    typedef cusp::csr_matrix<IndexType, ValueType, cusp::host_memory> HostMatrix;
    HostMatrix host_matrix;

    if (filename == "")
    {
        cusp::gallery::poisson5pt(host_matrix, 512, 512);
        std::cout << "Generated matrix (poisson5pt) ";
    }
    else
    {
        cusp::io::read_matrix_market_file(host_matrix, filename);
        std::cout << "Read matrix (" << filename << ") ";
    }
        
    std::cout << "with shape ("  << host_matrix.num_rows << "," << host_matrix.num_cols << ") and "
              << host_matrix.num_entries << " entries" << "\n\n";
    
    FILE * fid = fopen(BENCHMARK_OUTPUT_FILE_NAME, "a");
    fprintf(fid, "file=%s rows=%d cols=%d nonzeros=%d\n", filename.c_str(), (int) host_matrix.num_rows, (int) host_matrix.num_cols, (int) host_matrix.num_entries);
    fclose(fid);
    
    using namespace nitro;
    using namespace thrust;

    typedef tuple<HostMatrix *> ArgTuple;

    context cx;
    code_variant<tuning_policies::spmv, ArgTuple, sm> spmv(cx);

    coo_type<HostMatrix> coo;
    csr_type<HostMatrix> csr;
    csr_tex_type<HostMatrix> csr_tex;
    csr_vector_type<HostMatrix> csr_vector;
    csr_vector_tex_type<HostMatrix> csr_vector_tex;
    dia_type<HostMatrix> dia;
    dia_tex_type<HostMatrix> dia_tex;
    ell_type<HostMatrix> ell;
    ell_tex_type<HostMatrix> ell_tex;

    //spmv.add_variant(&coo);
    spmv.add_variant(&csr);
    //spmv.add_variant(&csr_tex);
    spmv.add_variant(&csr_vector);
    //spmv.add_variant(&csr_vector_tex);
    spmv.add_variant(&dia);
    //spmv.add_variant(&dia_tex);
    spmv.add_variant(&ell);
    //spmv.add_variant(&ell_tex);

    spmv.set_default(&csr_vector);

    avg_nz_per_row_type<HostMatrix> avg_nz_per_row;
    row_length_sd_type<HostMatrix> row_length_sd;
    nnz_type<HostMatrix> nnz;
    num_rows_type<HostMatrix> num_rows;
    max_deviation_type<HostMatrix> max_deviation;
    exceptional_rows_type<HostMatrix> exceptional_rows;
    dia_fillin_type<HostMatrix> dia_fillin;
    ell_fillin_type<HostMatrix> ell_fillin;

    spmv.add_input_feature(&avg_nz_per_row);
    spmv.add_input_feature(&row_length_sd);
    //spmv.add_input_feature(&nnz);
    //spmv.add_input_feature(&num_rows);
    spmv.add_input_feature(&max_deviation);
    //spmv.add_input_feature(&exceptional_rows);
    spmv.add_input_feature(&dia_fillin);
    spmv.add_input_feature(&ell_fillin);

    dia_cutoff_type dia_cutoff;
    ell_cutoff_type ell_cutoff;

    spmv.add_constraint(&dia, &dia_cutoff);
    spmv.add_constraint(&dia_tex, &dia_cutoff);
    spmv.add_constraint(&ell, &ell_cutoff);
    spmv.add_constraint(&ell_tex, &ell_cutoff);

    spmv(&host_matrix);
}

int main(int argc, char** argv)
{
    std::string filename = process_args(argc, argv);

    if (args.count("help"))
    {
        usage(argc, argv);
        return 0;
    }

    // select ValueType
    std::string value_type = args.count("value_type") ? args["value_type"] : "double";
    std::cout << "\nComputing SpMV with \'" << value_type << "\' values.\n\n";

    if(value_type == "float") {
      test_all_formats<int,float>(filename);
    } else if (value_type == "double")
    {
        int current_device = -1;
        hipDeviceProp_t properties;
        hipGetDevice(&current_device);
        hipGetDeviceProperties(&properties, current_device);
        if (properties.major == 1 && properties.minor < 3)
            std::cerr << "ERROR: Support for \'double\' requires Compute Capability 1.3 or greater\n\n";
        else
            test_all_formats<int,double>(filename);
    }
    else
    {
        std::cerr << "ERROR: Unsupported type \'" << value_type << "\'\n\n";
    }

    return 0;
}

